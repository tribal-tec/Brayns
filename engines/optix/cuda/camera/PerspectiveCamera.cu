#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "../Helpers.h"
#include "../Random.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

struct PerRayData_radiance
{
    float3 result;
    float importance;
    int depth;
};

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float, scene_epsilon, , );
rtBuffer<uchar4, 2> output_buffer;
rtBuffer<float4, 2> accum_buffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(float, aperture_radius, , );
rtDeclareVariable(float, focal_scale, , );
rtDeclareVariable(float4, jitter4, , );
rtDeclareVariable(unsigned int, samples_per_pixel, , );

rtBuffer<float4, 1> clip_planes;
rtDeclareVariable(unsigned int, nb_clip_planes, , );

rtDeclareVariable(uint, tonemapping, , );
rtDeclareVariable(float, mygamma, , );
rtDeclareVariable(float, white_point, , );
rtDeclareVariable(float, highlights, , );
rtDeclareVariable(float, blacks, , );
rtDeclareVariable(float, saturation, , );
rtDeclareVariable(float, brightness, , );

__device__ void getClippingValues(const float3& ray_origin,
                                  const float3& ray_direction, float& near,
                                  float& far)
{
    for (int i = 0; i < nb_clip_planes; ++i)
    {
        float4 clipPlane = clip_planes[i];
        const float3 planeNormal = {clipPlane.x, clipPlane.y, clipPlane.z};
        float rn = dot(ray_direction, planeNormal);
        if (rn == 0.f)
            rn = scene_epsilon;
        float d = clipPlane.w;
        float t = -(dot(planeNormal, ray_origin) + d) / rn;
        if (rn > 0.f) // opposite direction plane
            near = max(near, t);
        else
            far = min(far, t);
    }
}

// Pass 'seed' by reference to keep randomness state
__device__ float3 launch(unsigned int& seed, const float2 screen,
                         int i)
{
    const float2 offset[] = {
            make_float2(1/8.0f, 3/8.0f),
            make_float2(3/8.0f, 1/8.0f),
            make_float2(5/8.0f, 7/8.0f),
            make_float2(7/8.0f, 5/8.0f)
    };
    // Subpixel jitter: send the ray through a different position inside the
    // pixel each time, to provide antialiasing.
    const float2 subpixel_jitter = (frame+i) < 4 ? offset[i]
                       : make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f);

    const float2 d =
        (make_float2(launch_index) + subpixel_jitter) / screen * 2.f - 1.f;

    float3 ray_origin = eye;
    float3 ray_direction = normalize(d.x * U + d.y * V + W);

    float fs = focal_scale == 0.f ? 1.f : focal_scale;
    const float3 ray_target = ray_origin + fs * ray_direction;

    // lens sampling
    float2 sample = optix::square_to_disk(make_float2(jitter4.z, jitter4.w));

    ray_origin =
        ray_origin +
        aperture_radius * (sample.x * normalize(U) + sample.y * normalize(V));

    ray_direction = normalize(ray_target - ray_origin);

    float near = scene_epsilon;
    float far = INFINITY;

    getClippingValues(ray_origin, ray_direction, near, far);
    optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, near, far);

    PerRayData_radiance prd;
    prd.importance = 1.f;
    prd.depth = 0;

    rtTrace(top_object, ray, prd);

    return prd.result;
}

RT_PROGRAM void perspectiveCamera()
{
    const size_t2 screen = output_buffer.size();
    const float2 screen_f = make_float2(screen);

    unsigned int seed =
        tea<16>(screen.x * launch_index.y + launch_index.x, frame);

    const int num_samples = max(1, samples_per_pixel);

    float3 result = make_float3(0, 0, 0);
    for (int i = 0; i < num_samples; i++)
        result += launch(seed, screen_f, i);
    result /= num_samples;

    float4 acc_val;
    if (frame > 0)
        acc_val = lerp(accum_buffer[launch_index], make_float4(result, 0.f),
                       1.0f / static_cast<float>(frame + 1));
    else
        acc_val = make_float4(result, 1.f);

    float3 color = make_float3(acc_val);
    if(tonemapping != 0)
        color = tonemap2(color, mygamma, white_point, highlights, blacks, saturation, brightness);
    output_buffer[launch_index] = make_color(color);

    if(accum_buffer.size().x > 1 && accum_buffer.size().y > 1)
        accum_buffer[launch_index] = acc_val;
}

RT_PROGRAM void exception()
{
#if USE_DEBUG_EXCEPTIONS
    const unsigned int code = rtGetExceptionCode();
    rtPrintf("Exception code 0x%X at (%d, %d)\n", code, launch_index.x, launch_index.y);
#endif
    output_buffer[launch_index] = make_color(bad_color);
}
