#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <optix_world.h>
#include "../../CommonStructs.h"
#include "../Helpers.h"

using namespace optix;

//#define TEST

// Scene
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

// Material attributes

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Textures
#ifndef TEST
rtDeclareVariable(int, albedoMetallic_map, , );
rtDeclareVariable(int, normalRoughness_map, , );
rtDeclareVariable(float2, texcoord, attribute texcoord, );
#endif
rtDeclareVariable(float, metalness, , );
rtDeclareVariable(float, roughness, , );

// Lights
rtBuffer<BasicLight> lights;

// Vertices
rtDeclareVariable(float3, v0, attribute v0, );
rtDeclareVariable(float3, v1, attribute v1, );
rtDeclareVariable(float3, v2, attribute v2, );
rtDeclareVariable(float2, t0, attribute t0, );
rtDeclareVariable(float2, t1, attribute t1, );
rtDeclareVariable(float2, t2, attribute t2, );

rtDeclareVariable(float2, ddx, attribute ddx, );
rtDeclareVariable(float2, ddy, attribute ddy, );

rtDeclareVariable(uint, use_envmap, , );
rtDeclareVariable(int, envmap_radiance, , );
rtDeclareVariable(int, envmap_irradiance, , );
rtDeclareVariable(int, envmap_brdf_lut, , );
rtDeclareVariable(uint, radianceLODs, , );

static __device__ inline float calculateAttenuation(float3 WorldPos, float3 lightPos)
{
    float distance = length(lightPos - WorldPos);
    return 1.0f / (distance * distance);
}

static __device__ inline float distributionGGX(float3 N, float3 H, float roughness)
{
    const float a = roughness * roughness;
    const float a2 = a*a;
    const float NdotH  = max(dot(N, H), 0.0f);
    const float NdotH2 = NdotH * NdotH;

    float denom = (NdotH2 * (a2 - 1.0f) + 1.0f);
    denom = M_PIf * denom * denom;

    return a2 / denom;
}

static __device__ inline float GeometrySchlickGGX(float NdotV, float roughness)
{
    const float r = roughness + 1.0f;
    const float k = (r*r) / 8.0f;

    float denom = NdotV * (1.0 - k) + k;

    return NdotV / denom;
}

static __device__ inline float geometrySmith(float3 N, float3 V, float3 L, float roughness)
{
    const float NdotV = max(dot(N, V), 0.0f);
    const float NdotL = max(dot(N, L), 0.0f);
    const float ggx2  = GeometrySchlickGGX(NdotV, roughness);
    const float ggx1  = GeometrySchlickGGX(NdotL, roughness);

    return ggx1 * ggx2;
}

static __device__ inline float3 fresnelSchlick(float cosTheta, float3 F0)
{
    return F0 + (1.0f - F0) * pow(1.0f - cosTheta, 5.0f);
}

//static __device__ inline float3 fresnelSchlickRoughness(float cosTheta, float3 F0, float roughness)
//{
//    return F0 + (max(make_float3(1.0f - roughness), F0) - F0) * pow(1.0f - cosTheta, 5.0f);
//}

static __device__ inline float3 getIBLContribution(float NdV, float roughness, const float3& n, const float3& reflection, const float3& diffuseColor, const float3& specularColor)
{
    // Sample 2 levels and mix between to get smoother degradation
    const float ENV_LODS = 6.0f;
    float blend = roughness * ENV_LODS;
    float level0 = floor(blend);
    float level1 = min(ENV_LODS, level0 + 1.0f);
    blend -= level0;

    // Sample the specular env map atlas depending on the roughness value
    float2 uvSpec = getEquirectangularUV(reflection);
    uvSpec.y /= 2.0f;

    float2 uv0 = uvSpec;
    float2 uv1 = uvSpec;

    uv0 /= pow(2.0f, level0);
    uv0.y += 1.0f - exp(-M_LN2f * level0);

    uv1 /= pow(2.0f, level1);
    uv1.y += 1.0f - exp(-M_LN2f * level1);

    float2 irradianceUV = getEquirectangularUV(n);

    float3 diffuseLight = make_float3(RGBMToLinear(rtTex2D<float4>(envmap_irradiance, irradianceUV.x, 1.f-irradianceUV.y)));
    float3 specular0 = make_float3(RGBMToLinear(rtTex2D<float4>(envmap_radiance, uv0.x, 1.f-uv0.y)));
    float3 specular1 = make_float3(RGBMToLinear(rtTex2D<float4>(envmap_radiance, uv1.x, 1.f-uv1.y)));

    float3 specularLight = lerp(specular0, specular1, blend);
    float3 diffuse = diffuseLight * diffuseColor;

    const float3 brdf = make_float3(SRGBtoLinear(rtTex2D<float4>(envmap_brdf_lut, NdV, roughness)));

    // Bit of extra reflection for smooth materials
    float reflectivity = pow((1.0 - roughness), 2.0) * 0.05;
    float3 specular = specularLight * (specularColor * brdf.x + brdf.y + reflectivity);

    return diffuse + specular;
}


static __device__ inline void shade()
{
    const float3 world_shading_normal = optix::normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

    //float3 N = optix::faceforward(world_geometric_normal, -ray.direction,
    //                              world_geometric_normal);
    float3 N = world_shading_normal;

#ifndef TEST
    const float3 edge1 = v1 - v0;
    const float3 edge2 = v2 - v0;
    const float2 deltaUV1 = t1 - t0;
    const float2 deltaUV2 = t2 - t0;
    float f = 1.0f / (deltaUV1.x * deltaUV2.y - deltaUV2.x * deltaUV1.y);

    float3 tangent;
    tangent.x = f * (deltaUV2.y * edge1.x - deltaUV1.y * edge2.x);
    tangent.y = f * (deltaUV2.y * edge1.y - deltaUV1.y * edge2.y);
    tangent.z = f * (deltaUV2.y * edge1.z - deltaUV1.y * edge2.z);
    tangent = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, tangent));

    tangent = normalize(tangent - dot(tangent, N) * N);

    float3 bitangent = cross(N,tangent);
#endif

    const float3 WorldPos = ray.origin + t_hit * ray.direction;
    const float3 V = -ray.direction;

#ifdef TEST
    const float4 albedoMetallic = make_float4(1.f, 0.f, 0.f, metalness);
    const float3 albedo = make_float3(albedoMetallic);
    const float4 normalRoughness = make_float4(0.f, 0.f, 0.f, roughness);
#else
    const float4 albedoMetallic = SRGBtoLinear(rtTex2DGrad<float4>(albedoMetallic_map, texcoord.x, texcoord.y, ddx, ddy));
    const float3 albedo = make_float3(albedoMetallic);

    const float4 normalRoughness = rtTex2DGrad<float4>(normalRoughness_map, texcoord.x, texcoord.y, ddx, ddy);
    const float3 normal = normalize(make_float3(normalRoughness));

    optix::Matrix3x3 TBN;
    TBN.setCol(0,tangent);
    TBN.setCol(1,bitangent);
    TBN.setCol(2,N);

    N = normalize(TBN * (normal * 2.0f - 1.0f));
#endif

    const float3 F0 = lerp(make_float3(0.04f), albedo, albedoMetallic.w);

    float3 Lo = make_float3(0.0f);
    unsigned int num_lights = 0;//lights.size();
    for (int i = 0; i < num_lights; ++i)
    {
        // per-light radiance
        //const BasicLight& light = lights[i];
        BasicLight light = lights[i];
        light.pos = make_float3(0.5f, 1.0f, 1.5f); // Thisisshit !!
        const float3 L = normalize(light.pos - WorldPos);
        const float3 H = normalize(V + L);
        const float attenuation = calculateAttenuation(WorldPos, light.pos);
        const float3 radiance = light.color * attenuation * 20.0f; // 20.0f is shit !!!

        // cook-torrance brdf
        const float NDF = distributionGGX(N, H, normalRoughness.w);
        const float G = geometrySmith(N, V, L, normalRoughness.w);
        const float3 F = fresnelSchlick(max(dot(H, V), 0.0f), F0);

        const float3 kD = (make_float3(1.0f) - F) * (1.0f - albedoMetallic.w);

        const float3 numerator = NDF * G * F;
        const float NdotL = max(dot(N, L), 0.0f);
        const float denominator = 4.0f * max(dot(N, V), 0.0f) * NdotL;
        const float3 specular = numerator / max(denominator, 0.001f);

        Lo += (kD * albedo / M_PIf + specular) * radiance * NdotL;
    }

    float3 ambient = make_float3(0.03f) * albedo /* * ao*/;
    if (use_envmap)
    {
        float NdV = clamp(fabs(dot(N, V)), 0.001f, 1.0f);
        float3 reflection = normalize(reflect(-V, N));

        float3 f0 = make_float3(0.04);
        float3 diffuseColor = albedo * (make_float3(1.f) - f0) * (1.0 - albedoMetallic.w);
        float3 specularColor = lerp(f0, albedo, albedoMetallic.w);

        ambient = getIBLContribution(NdV, normalRoughness.w, N, reflection, diffuseColor, specularColor);
//        const float2 irradianceUV = getEquirectangularUV(N);
//        const float2 radianceUV = getEquirectangularUV(reflect(-V, N));

//        const float3 F = fresnelSchlickRoughness(max(dot(N, V), 0.0), F0, normalRoughness.w);
//        const float3 kD = (make_float3(1.0f) - F) * (1.0f - albedoMetallic.w);

//        float3 irradiance = make_float3(rtTex2D<float4>(envmap_irradiance, irradianceUV.x, irradianceUV.y));
//        //irradiance = pow(irradiance, 1.f/2.2f);
//        const float3 diffuse = irradiance * albedo;

//        // sample both the pre-filter map and the BRDF lut and combine them together as per the Split-Sum approximation to get the IBL specular part.
//        float3 prefilteredColor = make_float3(rtTex2DLod<float4>(envmap_radiance, radianceUV.x, radianceUV.y, normalRoughness.w * float(radianceLODs)));
//        //prefilteredColor = pow(prefilteredColor, 1.f/2.2f);
//        const float2 brdf = make_float2(rtTex2D<float4>(envmap_brdf_lut, max(dot(N, V), 0.0), normalRoughness.w));
//        const float3 specular = prefilteredColor * (F * brdf.x + brdf.y);

//        ambient = (kD * diffuse + specular)/* * ao*/;
    }

    const float3 color = ambient + Lo;
    prd.result = linearToSRGB(color / (color + make_float3(1.0f)));;
}

RT_PROGRAM void any_hit_shadow()
{
    rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{
    shade();
}
